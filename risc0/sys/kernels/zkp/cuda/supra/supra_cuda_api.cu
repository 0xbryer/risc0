#include "hip/hip_runtime.h"
#include <util/exception.cuh>
#include <util/gpu_t.cuh>
#include <util/rusterror.h>

#ifdef FEATURE_BABY_BEAR
#include <ff/baby_bear.hpp>
#endif

#include "calc_prefix_operation.cuh"
#include "poly_divide.cuh"
#include "poseidon_baby_bear/poseidon2.cu"

extern "C" RustError::by_value
sppark_poseidon2_fold(poseidon_out_t* d_out, const poseidon_in_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = num_hashes < 256 ? num_hashes : 256;
  size_t num_blocks = num_hashes < 256 ? 1 : num_hashes / 256;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon2_rows(poseidon_out_t* d_out, const fr_t* d_in, uint32_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

enum Operation { OpAdd, OpMultiply };

extern "C" RustError::by_value
sppark_calc_prefix_operation(Fp4* in_elems, uint32_t count, Operation op) {
  const gpu_t& gpu = select_gpu();

  auto device_props = gpu_props(gpu.id());
  int major = device_props.major;
  const uint32_t CHUNK_MULT_80 = 44;
  const uint32_t CHUNK_MULT_70 = 45;

  uint32_t block_size = 256;
  const uint32_t CHUNK_ADD = 48;
  uint32_t CHUNK_MULT = major >= 8 ? CHUNK_MULT_80 : CHUNK_MULT_70;
  uint32_t tmp_num_block_add = (count + (block_size * CHUNK_ADD) - 1) / (block_size * CHUNK_ADD);
  uint32_t tmp_num_block_mult = (count + (block_size * CHUNK_MULT) - 1) / (block_size * CHUNK_MULT);
  uint32_t sm_count = gpu.sm_count();
  uint32_t num_blocks_add = std::min(tmp_num_block_add, sm_count);
  uint32_t num_blocks_mult = std::min(tmp_num_block_mult, sm_count);
  size_t sharedMemorySize = block_size * sizeof(Fp4);

  try {
    dev_ptr_t<Fp4> d_mem(count, gpu);
    Fp4* d_elems = &d_mem[0];

    gpu.HtoD(d_elems, in_elems, count);

    if (op == Operation::OpAdd)
      gpu.launch_coop(calc_prefix_op_kernel<Add<Fp4>, Fp4, CHUNK_ADD>,
                      {num_blocks_add, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);
    else if (major >= 8)
      gpu.launch_coop(calc_prefix_op_kernel<Multiply<Fp4>, Fp4, CHUNK_MULT_80>,
                      {num_blocks_mult, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);
    else
      gpu.launch_coop(calc_prefix_op_kernel<Multiply<Fp4>, Fp4, CHUNK_MULT_70>,
                      {num_blocks_mult, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);

    gpu.DtoH(in_elems, d_elems, count);

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
poly_divide(fr4_t* polynomial, size_t poly_sz, fr4_t* remainder, const fr4_t* pow) {
  const gpu_t& gpu = select_gpu();
  uint32_t sm_count = gpu.sm_count();
  uint32_t sm_ceil = round_to_next_power_of_2(sm_count);

  const uint32_t divisor_degree = DIVISOR_DEGREE;
  const uint32_t block_sz = DIV_BLOCK_SZ;
  uint32_t nwarps_red = sm_ceil < WARP_SZ ? sm_ceil : WARP_SZ;
  uint32_t nwarps_sm = sm_ceil < WARP_SZ ? 1 : sm_ceil / WARP_SZ;

  uint32_t shared_powers_size = 3 * WARP_SZ / 2 + nwarps_red + nwarps_sm / 2;

  size_t shared_sz = sizeof(fr4_t) * ((block_sz / WARP_SZ) + shared_powers_size);

  size_t nthreads = sm_count * block_sz;
  size_t nloops = (poly_sz - divisor_degree + nthreads - 1) / nthreads;

  assert(block_sz >= sm_ceil);

  try {
    event_t sync_event;

    dev_ptr_t<fr4_t> d_mem(2 * sm_ceil + 2 * divisor_degree + poly_sz + block_sz + 1, gpu[0]);
    fr4_t* d_acc = &d_mem[0];
    fr4_t* d_prev_acc = &d_mem[sm_ceil];
    fr4_t* d_polynomial = &d_prev_acc[2 * divisor_degree];
    fr4_t(*_d_z_powers)[DIV_BLOCK_SZ] =
        reinterpret_cast<decltype(_d_z_powers)>(&d_polynomial[poly_sz]);

    sync_event.record(gpu[0]);

    generate_z_powers<<<1, block_sz, 0, gpu[0]>>>(_d_z_powers, pow[0], sm_ceil);

    sync_event.wait(gpu);
    sync_event.record(gpu[0]);

    gpu.HtoD(d_polynomial, polynomial, poly_sz);
    gpu.bzero(d_acc, sm_ceil + 2 * divisor_degree);

    sync_event.wait(gpu);

    const fr4_t(*d_z_powers)[DIV_BLOCK_SZ] = _d_z_powers;
    gpu.launch_coop(div_by_x_minus_z,
                    {sm_count, block_sz, shared_sz},
                    d_polynomial,
                    d_acc,
                    d_prev_acc,
                    poly_sz,
                    nloops,
                    d_z_powers);

    gpu.DtoH(remainder, &d_polynomial[0], 1);
    gpu.DtoH(polynomial, &d_polynomial[1], poly_sz - 1);

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}
